
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void mul_a(float a, float* x)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;
	x[i] *= a;
}

void cuda_mul_a(float a, float *x, int count)
{
	// Copy data to device.
	float* device_x;

	hipMalloc(&device_x, count * sizeof(float));
	hipMemcpy(device_x, x, count * sizeof(float), hipMemcpyHostToDevice);
	
	// Launch the kernel.
	mul_a<<<((count + 255) >> 8), 256>>>(a, device_x);
	
	// Copy data back to host.
	hipDeviceSynchronize();
	hipMemcpy(x, device_x, count * sizeof(float), hipMemcpyDeviceToHost);
	
	hipDeviceReset();
}
